/* Furthest point sampling GPU implementation
 * Author Zhaoyu SU
 * All Rights Reserved. Sep., 2019.
 * Happy Mid-Autumn Festival! :)
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#define USECPSEC 1000000ULL

__global__ void get_roi_bbox_gpu_kernel(int batch_size, int npoint, int nbbox, int bbox_attr,
                                            int diff_thres, int cls_thres, float expand_ratio,
                                            const float* input_coors,
                                            const float* gt_bbox,
                                            const int* input_num_list,
                                            const float* anchor_size,
                                            int* input_accu_list,
                                            float* roi_bbox,
                                            int* roi_conf,
                                            int* roi_diff) {
    if (batch_size * nbbox * bbox_attr <=0 || npoint <=0) {
//        printf("Get RoI Logits Op exited unexpectedly.\n");
        return;
    }
//    const float PI = 3.1415927;
    float anchor_diag = sqrtf(anchor_size[0]*anchor_size[0] + anchor_size[1]*anchor_size[1]);
    input_accu_list[0] = 0;
    for (int b=1; b<batch_size; b++) {
        input_accu_list[b] = input_accu_list[b-1] + input_num_list[b-1];
    }
    __syncthreads();
//    printf("%d\n", input_accu_list[5]);
    for (int b=blockIdx.x; b<batch_size; b+=gridDim.x) {
        for (int i=threadIdx.x; i<input_num_list[b]; i+=blockDim.x) {
            roi_bbox[input_accu_list[b]*7 + i*7 + 0] = 0.1;
            roi_bbox[input_accu_list[b]*7 + i*7 + 1] = 0.1;
            roi_bbox[input_accu_list[b]*7 + i*7 + 2] = 0.1;

            float point_x = input_coors[input_accu_list[b]*3 + i*3 + 0];
            float point_y = input_coors[input_accu_list[b]*3 + i*3 + 1];
            float point_z = input_coors[input_accu_list[b]*3 + i*3 + 2];
            roi_conf[input_accu_list[b] + i] = 0;
            roi_diff[input_accu_list[b] + i] = -1;
            for (int j=0; j<nbbox; j++) {
            // [w, l, h, x, y, z, r, cls, diff_idx]
            //  0  1  2  3  4  5  6   7      8
                float bbox_w = gt_bbox[b*nbbox*bbox_attr + j*bbox_attr + 0];
                float bbox_l = gt_bbox[b*nbbox*bbox_attr + j*bbox_attr + 1];
                float bbox_h = gt_bbox[b*nbbox*bbox_attr + j*bbox_attr + 2];
                float bbox_x = gt_bbox[b*nbbox*bbox_attr + j*bbox_attr + 3];
                float bbox_y = gt_bbox[b*nbbox*bbox_attr + j*bbox_attr + 4];
                float bbox_z = gt_bbox[b*nbbox*bbox_attr + j*bbox_attr + 5];
                float bbox_r = gt_bbox[b*nbbox*bbox_attr + j*bbox_attr + 6];
                float bbox_cls = gt_bbox[b*nbbox*bbox_attr + j*bbox_attr + 7];
                float bbox_diff = gt_bbox[b*nbbox*bbox_attr + j*bbox_attr + 8];
                if (bbox_l*bbox_h*bbox_w > 0) {
                    float rel_point_x = point_x - bbox_x;
                    float rel_point_y = point_y - bbox_y;
                    float rel_point_z = point_z - bbox_z;
                    float rot_rel_point_x = rel_point_x*cosf(bbox_r) + rel_point_y*sinf(bbox_r);
                    float rot_rel_point_y = -rel_point_x*sinf(bbox_r) + rel_point_y*cosf(bbox_r);
                    if (abs(rot_rel_point_x)<=bbox_w * (1 + expand_ratio) / 2 &&
                        abs(rot_rel_point_y)<=bbox_l * (1 + expand_ratio) / 2 &&
                        abs(rel_point_z)<=bbox_h * (1 + expand_ratio) / 2) {

                        roi_bbox[input_accu_list[b]*7 + i*7 + 0] = bbox_w;
                        roi_bbox[input_accu_list[b]*7 + i*7 + 1] = bbox_l;
                        roi_bbox[input_accu_list[b]*7 + i*7 + 2] = bbox_h;
                        roi_bbox[input_accu_list[b]*7 + i*7 + 3] = bbox_x;
                        roi_bbox[input_accu_list[b]*7 + i*7 + 4] = bbox_y;
                        roi_bbox[input_accu_list[b]*7 + i*7 + 5] = bbox_z;
                        roi_bbox[input_accu_list[b]*7 + i*7 + 6] = bbox_r;

//                        if (bbox_diff <= diff_thres && bbox_cls == 0) {
                        if (bbox_diff <= diff_thres && bbox_cls <= cls_thres) {
                            // Here we only take cars into consideration, while vans are excluded and give the foreground labels as -1 (ignored).
                            // TODO: need to change the category class accordingly to the expected detection target.
                            roi_conf[input_accu_list[b] + i] = 1;
                            roi_diff[input_accu_list[b] + i] = bbox_diff;
                        }else{
                            roi_conf[input_accu_list[b] + i] = -1;
                            roi_diff[input_accu_list[b] + i] = -1;
                        }
                    }
                }
            }
        }
    }
}

long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

void get_roi_bbox_gpu_launcher(int batch_size, int npoint, int nbbox, int bbox_attr,
                               int diff_thres, int cls_thres, float expand_ratio,
                               const float* input_coors,
                               const float* gt_bbox,
                               const int* input_num_list,
                               const float* anchor_size,
                               int* input_accu_list,
                               float* roi_bbox,
                               int* roi_conf,
                               int* roi_diff) {
    long long dt = dtime_usec(0);
    get_roi_bbox_gpu_kernel<<<32,512>>>(batch_size, npoint, nbbox, bbox_attr,
                                          diff_thres, cls_thres, expand_ratio,
                                          input_coors,
                                          gt_bbox,
                                          input_num_list,
                                          anchor_size,
                                          input_accu_list,
                                          roi_bbox,
                                          roi_conf,
                                          roi_diff);
    dt = dtime_usec(dt);
//	std::cout << "Voxel Sample (forward) CUDA time: " << dt/(float)USECPSEC << "s" << std::endl;
}
