/* 
3D IoU Calculation and Rotated NMS(modified from 2D NMS written by others)
Written by Shaoshuai Shi
All Rights Reserved 2018. 
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS_PER_BLOCK 16
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

//#define DEBUG
const int THREADS_PER_BLOCK_NMS = sizeof(unsigned long long) * 8; 
const float EPS = 1e-8;
struct Point {
    float x, y;
    __device__ Point() {}
    __device__ Point(float _x, float _y){
        x = _x, y = _y;
    }

    __device__ void set(float _x, float _y){
        x = _x; y = _y;
    }

    __device__ Point operator +(const Point &b)const{
        return Point(x + b.x, y + b.y);
    }

    __device__ Point operator -(const Point &b)const{
        return Point(x - b.x, y - b.y);
    }
};

__device__ inline float cross(const Point &a, const Point &b){
    return a.x * b.y - a.y * b.x;
}

__device__ inline float cross(const Point &p1, const Point &p2, const Point &p0){
    return (p1.x - p0.x) * (p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y);
}

__device__ int check_rect_cross(const Point &p1, const Point &p2, const Point &q1, const Point &q2){
    int ret = min(p1.x,p2.x) <= max(q1.x,q2.x)  && 
              min(q1.x,q2.x) <= max(p1.x,p2.x) &&
              min(p1.y,p2.y) <= max(q1.y,q2.y) &&
              min(q1.y,q2.y) <= max(p1.y,p2.y);
    return ret;
}

__device__ inline int check_in_box2d(const float *box, const Point &p){
    //params: box (5) [x1, y1, x2, y2, angle]
    const float MARGIN = 1e-5;

    float center_x = (box[0] + box[2]) / 2;
    float center_y = (box[1] + box[3]) / 2;
    float angle_cos = cos(box[4]), angle_sin = sin(box[4]);  // rotate the point in the opposite direction of box
    float rot_x = (p.x - center_x) * angle_cos + (p.y - center_y) * angle_sin + center_x;
    float rot_y = -(p.x - center_x) * angle_sin + (p.y - center_y) * angle_cos + center_y;
#ifdef DEBUG
    printf("box: (%.3f, %.3f, %.3f, %.3f, %.3f)\n", box[0], box[1], box[2], box[3], box[4]);
    printf("center: (%.3f, %.3f), cossin(%.3f, %.3f), src(%.3f, %.3f), rot(%.3f, %.3f)\n", center_x, center_y,
            angle_cos, angle_sin, p.x, p.y, (p.x - center_x) * angle_cos + (p.y - center_y) * angle_sin + center_x, rot_y);
#endif
    return (rot_x > box[0] - MARGIN && rot_x < box[2] + MARGIN && rot_y > box[1] - MARGIN && rot_y < box[3] + MARGIN);
}

__device__ inline int intersection(const Point &p1, const Point &p0, const Point &q1, const Point &q0, Point &ans){
    // fast exclusion 
    if (check_rect_cross(p0, p1, q0, q1) == 0) return 0;

    // check cross standing
    float s1 = cross(q0, p1, p0);
    float s2 = cross(p1, q1, p0);
    float s3 = cross(p0, q1, q0);
    float s4 = cross(q1, p1, q0);

    if (!(s1 * s2 > 0 && s3 * s4 > 0)) return 0;

    // calculate intersection of two lines
    float s5 = cross(q1, p1, p0);
    if(fabs(s5 - s1) > EPS){
        ans.x = (s5 * q0.x - s1 * q1.x) / (s5 - s1);
        ans.y = (s5 * q0.y - s1 * q1.y) / (s5 - s1);
    
    }
    else{
        float a0 = p0.y - p1.y, b0 = p1.x - p0.x, c0 = p0.x * p1.y - p1.x * p0.y;
        float a1 = q0.y - q1.y, b1 = q1.x - q0.x, c1 = q0.x * q1.y - q1.x * q0.y;
        float D = a0 * b1 - a1 * b0;

        ans.x = (b0 * c1 - b1 * c0) / D;
        ans.y = (a1 * c0 - a0 * c1) / D;
    }
    
    return 1;
}

__device__ inline void rotate_around_center(const Point &center, const float angle_cos, const float angle_sin, Point &p){
    float new_x = (p.x - center.x) * angle_cos + (p.y - center.y) * angle_sin + center.x;
    float new_y = -(p.x - center.x) * angle_sin + (p.y - center.y) * angle_cos + center.y;
    p.set(new_x, new_y);
}

__device__ inline int point_cmp(const Point &a, const Point &b, const Point &center){
    return atan2(a.y - center.y, a.x - center.x) > atan2(b.y - center.y, b.x - center.x);
}

__device__ inline float box_overlap(const float *box_a, const float *box_b){
    // params: box_a (5) [x1, y1, x2, y2, angle]
    // params: box_b (5) [x1, y1, x2, y2, angle]
    
    float a_x1 = box_a[0], a_y1 = box_a[1], a_x2 = box_a[2], a_y2 = box_a[3], a_angle = -box_a[4];
    float b_x1 = box_b[0], b_y1 = box_b[1], b_x2 = box_b[2], b_y2 = box_b[3], b_angle = -box_b[4];

    Point center_a((a_x1 + a_x2) / 2, (a_y1 + a_y2) / 2);
    Point center_b((b_x1 + b_x2) / 2, (b_y1 + b_y2) / 2);
#ifdef DEBUG
    printf("a: (%.3f, %.3f, %.3f, %.3f, %.3f), b: (%.3f, %.3f, %.3f, %.3f, %.3f)\n", a_x1, a_y1, a_x2, a_y2, a_angle,
           b_x1, b_y1, b_x2, b_y2, b_angle);
    printf("center a: (%.3f, %.3f), b: (%.3f, %.3f)\n", center_a.x, center_a.y, center_b.x, center_b.y);
#endif

    Point box_a_corners[5];
    box_a_corners[0].set(a_x1, a_y1);
    box_a_corners[1].set(a_x2, a_y1);
    box_a_corners[2].set(a_x2, a_y2);
    box_a_corners[3].set(a_x1, a_y2);

    Point box_b_corners[5];
    box_b_corners[0].set(b_x1, b_y1);
    box_b_corners[1].set(b_x2, b_y1);
    box_b_corners[2].set(b_x2, b_y2);
    box_b_corners[3].set(b_x1, b_y2);

    // get oriented corners 
    float a_angle_cos = cos(a_angle), a_angle_sin = sin(a_angle);
    float b_angle_cos = cos(b_angle), b_angle_sin = sin(b_angle);

    for (int k = 0; k < 4; k++){
#ifdef DEBUG
        printf("before corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k, box_a_corners[k].x, box_a_corners[k].y, box_b_corners[k].x, box_b_corners[k].y);
#endif
        rotate_around_center(center_a, a_angle_cos, a_angle_sin, box_a_corners[k]);
        rotate_around_center(center_b, b_angle_cos, b_angle_sin, box_b_corners[k]);
#ifdef DEBUG
        printf("corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k, box_a_corners[k].x, box_a_corners[k].y, box_b_corners[k].x, box_b_corners[k].y);
#endif
    }

    box_a_corners[4] = box_a_corners[0];
    box_b_corners[4] = box_b_corners[0];

    // get intersection of lines
    Point cross_points[16];
    Point poly_center;
    int cnt = 0, flag = 0;

    poly_center.set(0, 0);
    for (int i = 0; i < 4; i++){
        for (int j = 0; j < 4; j++){
            flag = intersection(box_a_corners[i + 1], box_a_corners[i], box_b_corners[j + 1], box_b_corners[j], cross_points[cnt]);
            if (flag){
                poly_center = poly_center + cross_points[cnt];
                cnt++;
            }
        }
    }

    // check corners
    for (int k = 0; k < 4; k++){
        if (check_in_box2d(box_a, box_b_corners[k])){
            poly_center = poly_center + box_b_corners[k];
            cross_points[cnt] = box_b_corners[k];
            cnt++;
        }
        if (check_in_box2d(box_b, box_a_corners[k])){
            poly_center = poly_center + box_a_corners[k];
            cross_points[cnt] = box_a_corners[k];
            cnt++;
        }
    }

    poly_center.x /= cnt;
    poly_center.y /= cnt;

    // sort the points of polygon
    Point temp;
    for (int j = 0; j < cnt - 1; j++){
        for (int i = 0; i < cnt - j - 1; i++){
            if (point_cmp(cross_points[i], cross_points[i + 1], poly_center)){
                temp = cross_points[i]; 
                cross_points[i] = cross_points[i + 1]; 
                cross_points[i + 1] = temp;
            }
        }
    }

#ifdef DEBUG
    printf("cnt=%d\n", cnt);
    for (int i = 0; i < cnt; i++){
        printf("All cross point %d: (%.3f, %.3f)\n", i, cross_points[i].x, cross_points[i].y);
    }
#endif

    // get the overlap areas
    float area = 0;
    for (int k = 0; k < cnt - 1; k++){
        area += cross(cross_points[k] - cross_points[0], cross_points[k + 1] - cross_points[0]);
    }

    return fabs(area) / 2.0;
}


// iou_bev => iou3d
__device__ inline float iou3d(const float *box_a, const float *box_b, const float* box_a_z, const float* box_b_z){
    // params: box_a (5) [x1, y1, x2, y2, angle]
    // params: box_b (5) [x1, y1, x2, y2, angle]
    // params: box_a_z (2) [z1min, z1max]
    // params: box_b_z (2) [z2min, z2max]
    float sa = (box_a[2] - box_a[0]) * (box_a[3] - box_a[1]);
    float sb = (box_b[2] - box_b[0]) * (box_b[3] - box_b[1]);
    float s_overlap = box_overlap(box_a, box_b);

    // compute the height
    float intersection_height = fminf(box_a_z[1], box_b_z[1]) - fmaxf(box_a_z[0], box_b_z[0]);

    //compute the volume
    float vol_a = sa * (box_a_z[1] - box_a_z[0]);
    float vol_b = sb * (box_b_z[1] - box_b_z[0]);
    float vol_overlap = s_overlap * intersection_height;
    
#ifdef DEBUG
    printf("sa, sb, s_overlap, vol_a, vol_b, vol_overlap: (%.3f, %.3f, %.3f, %.3f, %.3f, %.3f)\n", sa, sb, s_overlap, vol_a, vol_b, vol_overlap);
#endif

    return vol_overlap / fmaxf(vol_a + vol_b - vol_overlap, EPS);
}


__global__ void boxes_iou_3d_kernel(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_iou){
    // params: num_a : number of boxes in boxes_a
    // params: boxes_a (M, 7) [x, y, z, w, l, h, angle]
    // params: num_b : number of boxes in boxes_b
    // params: boxes_b (N, 7) [x, y, z, w, l, h, angle]
    const int a_idx = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
    const int b_idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    
    if (a_idx >= num_a || b_idx >= num_b){
        return;
    }

    const float * cur_box_a = boxes_a + a_idx * 7;
    const float * cur_box_b = boxes_b + b_idx * 7;

    float box_a_tmp[5];
    float box_b_tmp[5];
    float box_a_z_tmp[2];
    float box_b_z_tmp[2];


    // [x, y, z, w, l ,h, r]

    box_a_tmp[0] = cur_box_a[0] - cur_box_a[3] / 2; // x1,
    box_a_tmp[1] = cur_box_a[1] - cur_box_a[4] / 2; // y1
    box_a_tmp[2] = cur_box_a[0] + cur_box_a[3] / 2; // x2
    box_a_tmp[3] = cur_box_a[1] + cur_box_a[4] / 2; // y2
    box_a_tmp[4] = cur_box_a[6]; // ry

    box_a_z_tmp[0] = cur_box_a[2] - cur_box_a[5] / 2; // z1min
    box_a_z_tmp[1] = cur_box_a[2] + cur_box_a[5] / 2; // z1max


    box_b_tmp[0] = cur_box_b[0] - cur_box_b[3] / 2; // x1,
    box_b_tmp[1] = cur_box_b[1] - cur_box_b[4] / 2; // y1
    box_b_tmp[2] = cur_box_b[0] + cur_box_b[3] / 2; // x2
    box_b_tmp[3] = cur_box_b[1] + cur_box_b[4] / 2; // y2
    box_b_tmp[4] = cur_box_b[6]; // ry

    box_b_z_tmp[0] = cur_box_b[2] - cur_box_b[5] / 2; // z1min
    box_b_z_tmp[1] = cur_box_b[2] + cur_box_b[5] / 2; // z1max


    float cur_iou_3d = iou3d(&box_a_tmp[0], &box_b_tmp[0], &box_a_z_tmp[0], &box_b_z_tmp[0]);
    ans_iou[a_idx * num_b + b_idx] = cur_iou_3d;
}

__global__ void nms3d_kernel(const int boxes_num, const float nms_overlap_thresh,
                           const float *boxes, unsigned long long *mask){
    //params: boxes (N, 7) [x, y, z, w, l ,h, ry] z-up coordinate system
    //params: mask (N, N/THREADS_PER_BLOCK_NMS)

    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;

    // if (row_start > col_start) return;

    const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);
    const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);

    __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 7];

    if (threadIdx.x < col_size) {
        block_boxes[threadIdx.x * 7 + 0] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 0];
        block_boxes[threadIdx.x * 7 + 1] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 1];
        block_boxes[threadIdx.x * 7 + 2] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 2];
        block_boxes[threadIdx.x * 7 + 3] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 3];
        block_boxes[threadIdx.x * 7 + 4] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 4];
        block_boxes[threadIdx.x * 7 + 5] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 5];
        block_boxes[threadIdx.x * 7 + 6] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 6];
    }
    __syncthreads();

    if (threadIdx.x < row_size) {
        const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
        const float *cur_box = boxes + cur_box_idx * 7;

        float cur_box_tmp[5];
        float cur_box_z_tmp[2];

        cur_box_tmp[0] = cur_box[3] - cur_box[0] / 2; // x1,
        cur_box_tmp[1] = cur_box[4] - cur_box[1] / 2; // y1
        cur_box_tmp[2] = cur_box[3] + cur_box[0] / 2; // x2
        cur_box_tmp[3] = cur_box[4] + cur_box[1] / 2; // y2
        cur_box_tmp[4] = cur_box[6]; // ry

        cur_box_z_tmp[0] = cur_box[5] - cur_box[2] / 2; // z1min
        cur_box_z_tmp[1] = cur_box[5] + cur_box[2] / 2; // z1max


        int i = 0;
        unsigned long long t = 0;
        int start = 0;
        if (row_start == col_start) {
          start = threadIdx.x + 1;
        }
        for (i = start; i < col_size; i++) {
            const float *block_box_ptr = block_boxes + i * 7;

            float block_box_tmp[5];
            float block_box_z_tmp[2];
    
            block_box_tmp[0] = block_box_ptr[3] - block_box_ptr[0] / 2; // x1,
            block_box_tmp[1] = block_box_ptr[4] - block_box_ptr[1] / 2; // y1
            block_box_tmp[2] = block_box_ptr[3] + block_box_ptr[0] / 2; // x2
            block_box_tmp[3] = block_box_ptr[4] + block_box_ptr[1] / 2; // y2
            block_box_tmp[4] = block_box_ptr[6]; // ry
    
            block_box_z_tmp[0] = block_box_ptr[5] - block_box_ptr[2] / 2; // z1min
            block_box_z_tmp[1] = block_box_ptr[5] + block_box_ptr[2] / 2; // z1max

            float cur_iou_3d = iou3d(&cur_box_tmp[0], &block_box_tmp[0], &cur_box_z_tmp[0], &block_box_z_tmp[0]);

            if (cur_iou_3d > nms_overlap_thresh){
                t |= 1ULL << i;
            }
        }
        const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);
        mask[cur_box_idx * col_blocks + col_start] = t;
    }
}

void boxesIou3dGPUKernelLauncher(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_iou){

    dim3 blocks(DIVUP(num_b, THREADS_PER_BLOCK), DIVUP(num_a, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    boxes_iou_3d_kernel<<<blocks, threads>>>(num_a, boxes_a, num_b, boxes_b, ans_iou);
}


void nms3dGPUKernelLauncher(const int boxes_num, const float nms_overlap_thresh, const float *boxes, unsigned long long * mask ){
    dim3 blocks(DIVUP(boxes_num, THREADS_PER_BLOCK_NMS),
                DIVUP(boxes_num, THREADS_PER_BLOCK_NMS));
    dim3 threads(THREADS_PER_BLOCK_NMS);
    nms3d_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, boxes, mask);
}
