/* Voxel sampling GPU implementation
 * Author Zhaoyu SU
 * All Rights Reserved. Sep., 2019.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <float.h>  // import FLT_EPSILON

__global__ void voxel2col_gpu_kernel(int input_num, int channels, int input_voxel_size,
                                      int output_voxel_size, int kernel_size,
                                      const float* input_voxels,
                                      float* output_voxels,
                                      int* output_idx) {
//    const float PI = 3.1415927;
    const int input_voxel_num = input_voxel_size * input_voxel_size * input_voxel_size;
    const int output_voxel_num = output_voxel_size * output_voxel_size * output_voxel_size;
    const int kernel_num = kernel_size * kernel_size * kernel_size;
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < input_num * output_voxel_num * kernel_num) {
        output_idx[id] = -1;
    }
    __syncthreads();
    if (id < input_num * output_voxel_num * kernel_num) {
        int input_id = id / (output_voxel_num * kernel_num);
        int voxel_id = id % (input_id * kernel_num) / kernel_num;
        int kernel_id = id % kernel_num;

        int z = kernel_id / (kernel_size * kernel_size);
        int y = kernel_id % (kernel_size * kernel_size) / kernel_size;
        int x = kernel_id % kernel_size;

        int output_voxel_coor_z = voxel_id / (output_voxel_size * output_voxel_size);
        int output_voxel_coor_y = voxel_id % (output_voxel_size * output_voxel_size) / output_voxel_size;
        int output_voxel_coor_x = voxel_id % output_voxel_size;

        /*
        input_voxel_coor = output_voxel_coor + 1;
        kernel_coor = input_voxel_coor + [x/y/z] - 1, for x/y/z in [0, 1, 2];
        so:
        kernel_coor = output_voxel_coor + [x/y/z], for x/y/z in [0, 1, 2];
        */

        int kernel_coor_z = output_voxel_coor_z + z;
        int kernel_coor_y = output_voxel_coor_y + y;
        int kernel_coor_x = output_voxel_coor_x + x;
        int input_voxel_id = input_id * input_voxel_num + \
                             kernel_coor_z * input_voxel_size * input_voxel_size + \
                             kernel_coor_y * input_voxel_size + \
                             kernel_coor_x;
        int output_kernel_id = input_id * output_voxel_num * kernel_num + voxel_id * kernel_num + z * kernel_size * kernel_size + y * kernel_size + x;
        output_idx[output_kernel_id] = input_voxel_id;
        for (int c=0; c<channels; c++) {
            output_voxels[output_kernel_id * channels + c] = input_voxels[input_voxel_id * channels + c];
        }

    }
}



void voxel2col_gpu_launcher(int input_num, int channels, int input_voxel_size,
                             int output_voxel_size, int kernel_size,
                             const float* input_voxels,
                             float* output_voxels,
                             int* output_idx) {
    if (input_num*channels <=0 || input_voxel_size * output_voxel_size * kernel_size <= 0) {
        printf("DenseConvOp ERROR: Invalid CUDA input dimensions.\n");
        return;
    }
    const int output_voxel_num = output_voxel_size * output_voxel_size * output_voxel_size;
    const int kernel_num = kernel_size * kernel_size * kernel_size;
    int blockSize;      // The launch configurator returned block size
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize;       // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxel2col_gpu_kernel, 0, input_num * output_voxel_num * kernel_num);
    gridSize = (input_num * output_voxel_num * kernel_num + blockSize - 1) / blockSize;

    voxel2col_gpu_kernel<<<gridSize,blockSize>>>(input_num, channels, input_voxel_size,
                                                  output_voxel_size, kernel_size,
                                                  input_voxels,
                                                  output_voxels,
                                                  output_idx);
}
