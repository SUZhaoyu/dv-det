/* Voxel sampling GPU implementation
 * Author Zhaoyu SU
 * All Rights Reserved. Sep., 2019.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <float.h>

__device__ int binary_search(const long long* input_voxel_idx,
                             int start_id,
                             int stop_id,
                             long long target_voxel_id) {

    if (input_voxel_idx[start_id] > target_voxel_id || input_voxel_idx[stop_id] < target_voxel_id)
        return -1;
    while (start_id <= stop_id) {
        int m = start_id + (stop_id - start_id) / 2;
        if (input_voxel_idx[m] == target_voxel_id)
            return m;
        if (input_voxel_idx[m] < target_voxel_id)
            start_id = m + 1;
        else
            stop_id = m - 1;
    }
    return -1;
}

__device__ int get_batch_id(int* accu_list, int batch_size, int id) {
    for (int b=0; b<batch_size-1; b++) {
        if (id >= accu_list[b]) {
            if(id < accu_list[b+1])
                return b;
        }
    }
    return batch_size - 1;
}

__global__ void output_init_gpu_kernel(int batch_size, int center_num, int kernel_num,
                                       float padding, int channels,
                                       float* output_features,
                                       int* output_idx) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id < center_num * kernel_num) {
        output_idx[thread_id] = -1;
        for (int c=0; c<channels; c++) {
            output_features[thread_id*channels + c] = padding;
        }
    }
}

__global__ void grid_buffer_init_gpu_kernel(int batch_size, int input_point_num, float resolution,
                                            int grid_dim_w, int grid_dim_l, int grid_dim_h,
                                            const float* input_coors,
                                            int* input_accu_list,
                                            int* grid_buffer) {
    const int grid_dim_size = grid_dim_w * grid_dim_h * grid_dim_l;
    int point_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (point_id < input_point_num) {
        int center_grid_coor_x = (int)floor(input_coors[point_id*3 + 0] / resolution);
        int center_grid_coor_y = (int)floor(input_coors[point_id*3 + 1] / resolution);
        int center_grid_coor_z = (int)floor(input_coors[point_id*3 + 2] / resolution);
        int batch_id = get_batch_id(input_accu_list, batch_size, point_id);
        int grid_buffer_idx = batch_id * grid_dim_size + center_grid_coor_x * grid_dim_l * grid_dim_h + center_grid_coor_y * grid_dim_h + center_grid_coor_z;
        atomicExch(&grid_buffer[grid_buffer_idx], point_id);
    }
}


__global__ void voxel_sampling_gpu_kernel(int batch_size, int center_num, int channels,
                                          int kernel_size,
                                          int grid_dim_w, int grid_dim_l, int grid_dim_h,
                                          float resolution,
                                          const float* input_coors,
                                          const float* input_features,
                                          const float* center_coors,
                                          int* center_accu_list,
                                          int* grid_buffer,
                                          float* output_features,
                                          int* output_idx) {

	const int kernel_num = kernel_size * kernel_size * kernel_size;
	const int half_kernel_size = (kernel_size - 1) / 2;
	const int half_kernel_num = kernel_size * kernel_size * half_kernel_size + \
                                kernel_size * half_kernel_size + \
                                half_kernel_size;
	const int search_kernel_size = kernel_size + 1;
	const int search_kernel_num = search_kernel_size * search_kernel_size * search_kernel_size;
    const int grid_dim_size = grid_dim_w * grid_dim_l * grid_dim_h;
	const float radius = 1.5 * resolution;
	const float r2 = radius * radius;

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id < center_num * search_kernel_num) {

        int center_id = thread_id / search_kernel_num;
        int search_grid_id = thread_id % search_kernel_num;
        int batch_id = get_batch_id(center_accu_list, batch_size, center_id);

        float center_coor_x = center_coors[center_id*3 + 0];
        float center_coor_y = center_coors[center_id*3 + 1];
        float center_coor_z = center_coors[center_id*3 + 2];
        int center_grid_coor_x = __float2int_rz(center_coor_x / resolution);
        int center_grid_coor_y = __float2int_rz(center_coor_y / resolution);
        int center_grid_coor_z = __float2int_rz(center_coor_z / resolution);


        int search_grid_x = search_grid_id / (search_kernel_size * search_kernel_size);
        int search_grid_y = search_grid_id % (search_kernel_size * search_kernel_size) / search_kernel_size;
        int search_grid_z = search_grid_id % search_kernel_size;

        int search_offset_x = -2 + round(center_coor_x / resolution - center_grid_coor_x) + search_grid_x;
        int search_offset_y = -2 + round(center_coor_y / resolution - center_grid_coor_y) + search_grid_y;
        int search_offset_z = -2 + round(center_coor_z / resolution - center_grid_coor_z) + search_grid_z;

        int target_grid_x = max(0, min(center_grid_coor_x + search_offset_x, grid_dim_w - 1));
        int target_grid_y = max(0, min(center_grid_coor_y + search_offset_y, grid_dim_l - 1));
        int target_grid_z = max(0, min(center_grid_coor_z + search_offset_z, grid_dim_h - 1));
        int target_grid_id = batch_id * grid_dim_size + target_grid_x * grid_dim_l * grid_dim_h + target_grid_y * grid_dim_h + target_grid_z;
        int point_id = grid_buffer[target_grid_id];

        if (point_id>=0) {
            float coor_x = input_coors[point_id*3 +0];
            float coor_y = input_coors[point_id*3 +1];
            float coor_z = input_coors[point_id*3 +2];
            float dx = coor_x - center_coor_x + FLT_EPSILON;
            float dy = coor_y - center_coor_y + FLT_EPSILON;
            float dz = coor_z - center_coor_z + FLT_EPSILON;
            float dx2 = dx * dx;
            float dy2 = dy * dy;
            float dz2 = dz * dz;
            if (dx2 < r2 && dy2 < r2 && dz2 < r2) {
                int kernel_coor_x = __float2int_rz(dx / resolution + 0.5 * fabsf(dx) / dx);
                int kernel_coor_y = __float2int_rz(dy / resolution + 0.5 * fabsf(dy) / dy);
                int kernel_coor_z = __float2int_rz(dz / resolution + 0.5 * fabsf(dz) / dz);
                int voxel_coor = center_id * kernel_num + half_kernel_num + \
                                 kernel_size * kernel_size * kernel_coor_x + \
                                 kernel_size * kernel_coor_y + \
                                 kernel_coor_z;
                if (output_idx[voxel_coor] < 0) {
                    output_idx[voxel_coor] = point_id;
                    for (int c=0; c<channels; c++) {
                        output_features[voxel_coor * channels + c] = input_features[point_id * channels + c];
                    }
                }
            }
        }
	}
}


__global__ void voxel_sampling_grad_gpu_kernel(int kernel_number, int ngrid, int channels,
                                               const int* output_idx,
                                               const float* output_features_grad,
                                               float* input_features_grad) {
    int center_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (center_id < kernel_number) {
        for (int i=0; i<ngrid; i++) {
            int voxel_coor = center_id*ngrid + i;
            int point_id = output_idx[voxel_coor];
            if (point_id >= 0) {
                for (int c=0; c<channels; c++) {
                    atomicAdd(&input_features_grad[point_id*channels + c], output_features_grad[voxel_coor*channels + c]);
                }
            }
        }
    }
}


void voxel_sampling_gpu_launcher(int batch_size, int input_point_num, int channels,
                                 int center_num, int kernel_size,
                                 int grid_dim_w, int grid_dim_l, int grid_dim_h,
                                 float resolution, float padding,
                                 const float* input_coors,
                                 const float* input_features,
                                 const int* input_num_list,
                                 const float* center_coors,
                                 const int* center_num_list,
                                 int* input_accu_list,
                                 int* center_accu_list,
                                 int* grid_buffer,
                                 float* output_features,
                                 int* output_idx) {
    if (batch_size*input_point_num <=0 || center_num * channels <= 0) {
        printf("VoxelSampleOp ERROR: Invalid CUDA input dimensions.\n");
        return;
    }
    int kernel_num = kernel_size * kernel_size * kernel_size;
    int search_kernel_num = (kernel_size + 1) * (kernel_size + 1) * (kernel_size + 1);

    int blockSize;      // The launch configurator returned block size
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize;       // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, output_init_gpu_kernel, 0, center_num * kernel_num);
    gridSize = (center_num * kernel_num + blockSize - 1) / blockSize;
    output_init_gpu_kernel<<<gridSize, blockSize>>>(batch_size, center_num, kernel_num,
                                                    padding, channels,
                                                    output_features,
                                                    output_idx);

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, grid_buffer_init_gpu_kernel, 0, input_point_num);
    gridSize = (input_point_num + blockSize - 1) / blockSize;
    grid_buffer_init_gpu_kernel<<<gridSize, blockSize>>>(batch_size, input_point_num, resolution,
                                                         grid_dim_w, grid_dim_l, grid_dim_h,
                                                         input_coors,
                                                         input_accu_list,
                                                         grid_buffer);

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxel_sampling_gpu_kernel, 0, center_num * search_kernel_num);
    gridSize = (center_num * search_kernel_num + blockSize - 1) / blockSize;
    voxel_sampling_gpu_kernel<<<gridSize, blockSize>>>(batch_size, center_num, channels,
                                                       kernel_size,
                                                       grid_dim_w, grid_dim_l, grid_dim_h, resolution,
                                                       input_coors,
                                                       input_features,
                                                       center_coors,
                                                       center_accu_list,
                                                       grid_buffer,
                                                       output_features,
                                                       output_idx);
}


void voxel_sampling_grad_gpu_launcher(int kernel_number, int ngrid, int channels,
                                    const int* output_idx,
                                    const float* output_features_grad,
                                    float* input_features_grad) {
    if (kernel_number==0 || ngrid*channels == 0) {
        printf("VoxelSampleGradOp ERROR: Invalid CUDA input dimensions.\n");
        return;
    }
    int blockSize;      // The launch configurator returned block size
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize;       // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, output_init_gpu_kernel, 0, kernel_number);
    gridSize = (kernel_number + blockSize - 1) / blockSize;
    voxel_sampling_grad_gpu_kernel<<<gridSize, blockSize>>>(kernel_number, ngrid, channels,
                                                            output_idx,
                                                            output_features_grad,
                                                            input_features_grad);
}