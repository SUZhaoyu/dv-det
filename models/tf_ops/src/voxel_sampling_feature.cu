/* Voxel sampling GPU implementation
 * Author Zhaoyu SU
 * All Rights Reserved. Sep., 2019.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <float.h>

//__global__ void output_init_gpu_kernel(int center_num, int kernel_num,
//                                       float padding, int channels,
//                                       float* output_features) {
//    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
//    if (thread_id < center_num * kernel_num) {
//        for (int c=0; c<channels; c++) {
//            output_features[thread_id*channels + c] = padding;
//        }
//    }
//}


__global__ void voxel_sampling_feature_gpu_kernel(int center_num, int channels, int kernel_num, float padding,
                                                  int output_pooling_size,
                                                  const float* input_features,
                                                  const int* output_idx,
                                                  float* output_features) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int c = thread_id % channels;
    int voxel_coor = thread_id / channels;
    if (thread_id < center_num * kernel_num * channels) {
        int pooling_count = 0;
        for (int p=0; p<output_pooling_size; p++) {
            int point_id = output_idx[voxel_coor*output_pooling_size + p];
            if (point_id >= 0) {
                output_features[thread_id] += input_features[point_id * channels + c];
                pooling_count += 1;
//                printf("%f\n", input_features[point_id * channels + c]);
            }
        }
//        if (pooling_count > 1)
//            printf("%f\n", output_features[thread_id]);
        if (pooling_count > 0)
            output_features[thread_id] /= pooling_count;
        if (pooling_count == 0)
            output_features[thread_id] = padding;
	}
}


__global__ void voxel_sampling_feature_grad_gpu_kernel(int center_num, int kernel_num, int channels,
                                                       int output_pooling_size,
                                                       const int* output_idx,
                                                       const float* output_features_grad,
                                                       float* input_features_grad) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id < center_num * kernel_num) {
        int pooling_count = 0;
        for (int p=0; p<output_pooling_size; p++) {
            if (output_idx[thread_id*output_pooling_size + p] >= 0)
                pooling_count += 1;
        }

        for (int p=0; p<pooling_count; p++) {
            int point_id = output_idx[thread_id*output_pooling_size + p];
            for (int c=0; c<channels; c++) {
                atomicAdd(&input_features_grad[point_id*channels + c], output_features_grad[thread_id*channels + c] / pooling_count);
            }
        }
    }
}


void voxel_sampling_feature_gpu_launcher(int center_num, int kernel_num, int channels, float padding,
                                         int output_pooling_size,
                                         const float* input_features,
                                         const int* output_idx,
                                         float* output_features) {
    if (center_num * channels <= 0) {
        printf("VoxelSampleFeatureOp ERROR: Invalid CUDA input dimensions.\n");
        return;
    }

    int blockSize;      // The launch configurator returned block size
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize;       // The actual grid size needed, based on input size

//    cudaOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, output_init_gpu_kernel, 0, center_num * kernel_num);
//    gridSize = (center_num * kernel_num + blockSize - 1) / blockSize;
//    output_init_gpu_kernel<<<gridSize, blockSize>>>(center_num, kernel_num,
//                                                    padding, channels,
//                                                    output_features);

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxel_sampling_feature_gpu_kernel, 0, center_num * kernel_num * channels);
    gridSize = (center_num * kernel_num * channels + blockSize - 1) / blockSize;
    voxel_sampling_feature_gpu_kernel<<<gridSize, blockSize>>>(center_num, channels, kernel_num, padding,
                                                               output_pooling_size,
                                                               input_features,
                                                               output_idx,
                                                               output_features);
}


void voxel_sampling_feature_grad_gpu_launcher(int center_num, int kernel_num, int channels,
                                              int output_pooling_size,
                                              const int* output_idx,
                                              const float* output_features_grad,
                                              float* input_features_grad) {
    if (center_num==0 || kernel_num*channels == 0) {
        printf("VoxelSampleGradOp ERROR: Invalid CUDA input dimensions.\n");
        return;
    }
    int blockSize;      // The launch configurator returned block size
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize;       // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxel_sampling_feature_grad_gpu_kernel, 0, center_num * kernel_num);
    gridSize = (center_num * kernel_num + blockSize - 1) / blockSize;
    voxel_sampling_feature_grad_gpu_kernel<<<gridSize, blockSize>>>(center_num, kernel_num, channels,
                                                                    output_pooling_size,
                                                                    output_idx,
                                                                    output_features_grad,
                                                                    input_features_grad);
}